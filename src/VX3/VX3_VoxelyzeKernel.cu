#include "hip/hip_runtime.h"
#include "VX3_MemoryCleaner.h"
#include "VX3_VoxelyzeKernel.cuh"

/* Tools */
__device__ int bound(int x, int min, int max) {
    if (x < min)
        return min;
    if (x > max)
        return max;
    return x;
}

/* Sub GPU Threads */
__global__ void gpu_update_links(VX3_Link **links, int num);
__global__ void gpu_update_voxels(VX3_Voxel *voxels, int num, double dt, double currentTime, VX3_VoxelyzeKernel *k);
__global__ void gpu_update_temperature(VX3_Voxel *voxels, int num, double TempAmplitude, double TempPeriod, double currentTime, VX3_VoxelyzeKernel* k);
__global__ void gpu_update_attach(VX3_Voxel **surface_voxels, int num, double watchDistance, VX3_VoxelyzeKernel *k);
__global__ void gpu_update_cilia_force(VX3_Voxel **surface_voxels, int num, VX3_VoxelyzeKernel *k);
__global__ void gpu_update_occlusion(VX3_Voxel *voxels, VX3_Voxel **surface_voxels, int num, VX3_VoxelyzeKernel *k, bool surfVoxOnly, int lightOn);  // sam
__global__ void gpu_clear_lookupgrid(VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid, int num);
__global__ void gpu_insert_lookupgrid(VX3_Voxel **d_surface_voxels, int num, VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid,
                                      VX3_Vec3D<> *gridLowerBound, VX3_Vec3D<> *gridDelta, int lookupGrid_n);
__global__ void gpu_collision_attachment_lookupgrid(VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid, int num, double watchDistance,
                                                    VX3_VoxelyzeKernel *k);
__global__ void gpu_update_detach(VX3_Link **links, int num, VX3_VoxelyzeKernel *k);
__global__ void gpu_update_voxel_detachment(VX3_Voxel *voxels, VX3_Voxel **surface_voxels, int num, VX3_VoxelyzeKernel *k, bool surfVoxOnly); //sam
/* Host methods */

VX3_VoxelyzeKernel::VX3_VoxelyzeKernel(CVX_Sim *In) {

    voxSize = In->Vx.voxSize;

    num_d_voxelMats = In->Vx.voxelMats.size();
    VcudaMalloc((void **)&d_voxelMats, num_d_voxelMats * sizeof(VX3_MaterialVoxel));
    {
        // push all h first, since there will be reference below
        for (auto mat : In->Vx.voxelMats) {
            h_voxelMats.push_back(mat);
        }
        int i = 0;
        for (auto mat : In->Vx.voxelMats) {
            VX3_MaterialVoxel tmp_voxelMat(mat, this);
            VcudaMemcpy(d_voxelMats + i, &tmp_voxelMat, sizeof(VX3_MaterialVoxel), VcudaMemcpyHostToDevice);
            i++;
        }
    }

    num_d_linkMats = In->Vx.linkMats.size();
    VcudaMalloc((void **)&d_linkMats, num_d_linkMats * sizeof(VX3_MaterialLink));
    {
        int i = 0;
        std::vector<VX3_MaterialLink *> tmp_v_linkMats;
        for (CVX_MaterialLink *mat : In->Vx.linkMats) {
            // printf("mat->vox1Mat %p, mat->vox2Mat %p.\n", mat->vox1Mat,
            // mat->vox2Mat);
            VX3_MaterialLink tmp_linkMat(mat, this);
            VcudaMemcpy(d_linkMats + i, &tmp_linkMat, sizeof(VX3_MaterialLink), VcudaMemcpyHostToDevice);
            tmp_v_linkMats.push_back(d_linkMats + i);
            h_linkMats.push_back(mat);
            i++;
        }
        hd_v_linkMats = VX3_hdVector<VX3_MaterialLink *>(tmp_v_linkMats);
    }

    num_d_voxels = In->Vx.voxelsList.size();
    VcudaMalloc((void **)&d_voxels, num_d_voxels * sizeof(VX3_Voxel));
    for (int i = 0; i < num_d_voxels; i++) {
        h_voxels.push_back(In->Vx.voxelsList[i]);
        h_lookup_voxels[In->Vx.voxelsList[i]] = d_voxels + i;
    }
    VcudaMalloc((void **)&d_initialPosition, num_d_voxels * sizeof(Vec3D<>));

    num_d_links = In->Vx.linksList.size();
    std::vector<VX3_Link *> tmp_v_links;
    VcudaMalloc((void **)&d_links, num_d_links * sizeof(VX3_Link));
    VX3_Link *tmp_link_cache = (VX3_Link *)malloc(num_d_links * sizeof(VX3_Link));
    for (int i = 0; i < num_d_links; i++) {
        VX3_Link tmp_link(In->Vx.linksList[i], this);
        memcpy(tmp_link_cache + i, &tmp_link, sizeof(VX3_Link));
        tmp_v_links.push_back(d_links + i); // not copied yet, but still ok to get the address
        h_links.push_back(In->Vx.linksList[i]);
    }
    VcudaMemcpy(d_links, tmp_link_cache, num_d_links * sizeof(VX3_Link), VcudaMemcpyHostToDevice);
    hd_v_links = VX3_hdVector<VX3_Link *>(tmp_v_links);
    for (int i = 0; i < num_d_links; i++) {
        h_lookup_links[In->Vx.linksList[i]] = d_links + i;
    }

    for (int i = 0; i < num_d_voxels; i++) {
        // set values for GPU memory space
        VX3_Voxel tmp_voxel(In->Vx.voxelsList[i], this);
        VcudaMemcpy(d_voxels + i, &tmp_voxel, sizeof(VX3_Voxel), VcudaMemcpyHostToDevice);
    }

    // Not all data is in Vx, here are others:
    DtFrac = In->DtFrac;
    StopConditionType = In->StopConditionType;
    StopConditionValue = In->StopConditionValue;
    TempEnabled = In->pEnv->TempEnabled;
    VaryTempEnabled = In->pEnv->VaryTempEnabled;
    TempBase = In->pEnv->TempBase;
    TempAmplitude = In->pEnv->TempAmplitude;
    TempPeriod = In->pEnv->TempPeriod;
    // currentTemperature = TempBase + TempAmplitude;

    d_surface_voxels = NULL;
}

void VX3_VoxelyzeKernel::cleanup() {
    // The reason not use ~VX3_VoxelyzeKernel is that will be automatically call
    // multiple times after we use memcpy to clone objects.
    MycudaFree(d_linkMats);
    MycudaFree(d_voxels);
    MycudaFree(d_links);
    // MycudaFree(d_collisionsStale);
    if (d_surface_voxels) {
        MycudaFree(d_surface_voxels); // can __device__ malloc pointer be freed
                                      // by hipFree in __host__??
    }
    // MycudaFree(d_collisions);
}

/* Cuda methods : cannot use any CVX_xxx, and no std::, no boost::, and no
 * filesystem. */

__device__ void VX3_VoxelyzeKernel::syncVectors() {
    d_v_linkMats.clear();
    d_v_collisions.clear();
    d_targets.clear();
    // allocate memory for collision lookup table
    num_lookupGrids = lookupGrid_n * lookupGrid_n * lookupGrid_n;
    d_collisionLookupGrid = (VX3_dVector<VX3_Voxel *> *)malloc(num_lookupGrids * sizeof(VX3_dVector<VX3_Voxel *>));
    if (d_collisionLookupGrid == NULL) {
        printf(COLORCODE_BOLD_RED "ERROR: not enough memory.\n");
    }
    for (int i = 0; i < hd_v_linkMats.size(); i++) {
        d_v_linkMats.push_back(hd_v_linkMats[i]);
    }

    d_v_links.clear();
    for (int i = 0; i < hd_v_links.size(); i++) {
        d_v_links.push_back(hd_v_links[i]);
    }

    for (int i = 0; i < num_d_voxelMats; i++) {
        d_voxelMats[i].syncVectors();
    }

    for (int i = 0; i < num_d_linkMats; i++) {
        d_linkMats[i].syncVectors();
    }

    for (int i = 0; i < num_d_voxels; i++) {
        d_voxels[i].syncVectors();
    }
}
__device__ void VX3_VoxelyzeKernel::saveInitialPosition() {
    for (int i = 0; i < num_d_voxels; i++) {
        d_initialPosition[i] = d_voxels[i].pos;
        // Save this value to voxel, so it can be read out when collecting results in cpu.
        d_voxels[i].isMeasured = (bool) d_voxels[i].mat->isMeasured;
    }
}
__device__ bool VX3_VoxelyzeKernel::StopConditionMet(void) // have we met the stop condition yet?
{
    if (VX3_MathTree::eval(currentCenterOfMass.x, currentCenterOfMass.y, currentCenterOfMass.z, collisionCount, currentTime, recentAngle,
                           targetCloseness, numClosePairs, num_d_voxels, StopConditionFormula) > 0) {
        // double a =
        //     VX3_MathTree::eval(currentCenterOfMass.x, currentCenterOfMass.y, currentCenterOfMass.z, collisionCount, currentTime,
        //     StopConditionFormula);
        // printf("stop score: %f.\n\n", a);
        return true;
    }
    if (currentTime > 0 && num_d_surface_voxels <= 2)
        return true;
    if (forceExit)
        return true;
    return false;
    // if (StopConditionType != SC_MAX_SIM_TIME) {
    //     printf(COLORCODE_BOLD_RED "StopConditionType: %d. Type of stop condition no supported for "
    //                               "now.\n" COLORCODE_RESET,
    //            StopConditionType);
    //     return true;
    // }
    // return currentTime > StopConditionValue ? true : false;
}

__device__ double VX3_VoxelyzeKernel::recommendedTimeStep() {
    // find the largest natural frequency (sqrt(k/m)) that anything in the
    // simulation will experience, then multiply by 2*pi and invert to get the
    // optimally largest timestep that should retain stability
    double MaxFreq2 = 0.0f; // maximum frequency in the simulation in rad/sec
    if (!num_d_links) {
        printf("WARNING: No links.\n");
    }
    if (!num_d_voxels) {
        printf(COLORCODE_BOLD_RED "ERROR: No voxels.\n");
    }
    for (int i = 0; i < num_d_links; i++) {
        VX3_Link *pL = d_links + i;
        // axial
        double m1 = pL->pVNeg->mat->mass(), m2 = pL->pVPos->mat->mass();
        double thisMaxFreq2 = pL->axialStiffness() / (m1 < m2 ? m1 : m2);
        if (thisMaxFreq2 > MaxFreq2)
            MaxFreq2 = thisMaxFreq2;
        // rotational will always be less than or equal
    }
    if (MaxFreq2 <= 0.0f) {                      // didn't find anything (i.e no links) check for
                                                 // individual voxelss
        for (int i = 0; i < num_d_voxels; i++) { // for each link
            double thisMaxFreq2 = d_voxels[i].mat->youngsModulus() * d_voxels[i].mat->nomSize / d_voxels[i].mat->mass();
            if (thisMaxFreq2 > MaxFreq2)
                MaxFreq2 = thisMaxFreq2;
        }
    }
    if (MaxFreq2 <= 0.0f)
        return 0.0f;
    else
        return 1.0f / (6.283185f * sqrt(MaxFreq2)); // the optimal timestep is to advance one
                                                    // radian of the highest natural frequency
}

__device__ void VX3_VoxelyzeKernel::updateTemperature() {
    // updates the temperatures For Actuation!
    // different temperatures in different objs are not support for now.
    if (VaryTempEnabled) {
        if (TempPeriod > 0) {
            int blockSize = 512;
            int minGridSize;
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_temperature, 0,
                                               num_d_voxels); // Dynamically calculate blockSize
            int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize;
            int blockSize_voxels = num_d_voxels < blockSize ? num_d_voxels : blockSize;
            gpu_update_temperature<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, num_d_voxels, TempAmplitude, TempPeriod, currentTime, this);
            CUDA_CHECK_AFTER_CALL();
            VcudaDeviceSynchronize();
        }
    }
}

__device__ bool VX3_VoxelyzeKernel::doTimeStep(float dt) {
    // clock_t time_measures[10];
    // time_measures[0] = clock();
    updateTemperature();
    CurStepCount++;
    if (dt == 0)
        return true;
    else if (dt < 0) {
        if (!OptimalDt) {
            OptimalDt = recommendedTimeStep();
        }
        if (OptimalDt < 1e-10) {
            CUDA_DEBUG_LINE("recommendedTimeStep is zero.");
            OptimalDt = 1e-10;
            // return false;
        }
        dt = DtFrac * OptimalDt;
    }
    bool Diverged = false;

    int blockSize;
    int minGridSize;
    if (d_v_links.size()) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_links, 0,
                                           d_v_links.size()); // Dynamically calculate blockSize
        int gridSize_links = (d_v_links.size() + blockSize - 1) / blockSize;
        int blockSize_links = d_v_links.size() < blockSize ? d_v_links.size() : blockSize;
        // if (CurStepCount % 1000 == 0 || currentTime>1.0) {
        //     printf("&d_v_links[0] %p; d_v_links.size() %d. \n", &d_v_links[0], d_v_links.size());
        // }
        gpu_update_links<<<gridSize_links, blockSize_links>>>(&d_v_links[0], d_v_links.size());
        CUDA_CHECK_AFTER_CALL();
        VcudaDeviceSynchronize();

        // checking every link for diverge is too wasteful! using random
        // sampling.
        int r = random(d_v_links.size(), clock());
        if (d_v_links[r]->axialStrain() > 100) {
            CUDA_DEBUG_LINE("Diverged.");
            Diverged = true; // catch divergent condition! (if any thread sets
                             // true we will fail, so don't need mutex...
        }
        if (Diverged)
            return false;
    }

    if (isSurfaceChanged) {
        isSurfaceChanged = false;

        regenerateSurfaceVoxels();
    }

    if (enableAttach || EnableCollision) { // either attachment and collision need measurement for pairwise distances
        updateAttach();
    }
    if (enableDetach) {
        updateDetach();
    }

    if (EnableCilia) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_cilia_force, 0,
                                           num_d_surface_voxels); // Dynamically calculate blockSize
        int gridSize_voxels = (num_d_surface_voxels + blockSize - 1) / blockSize;
        int blockSize_voxels = num_d_surface_voxels < blockSize ? num_d_surface_voxels : blockSize;
        gpu_update_cilia_force<<<gridSize_voxels, blockSize_voxels>>>(d_surface_voxels, num_d_surface_voxels, this);
        CUDA_CHECK_AFTER_CALL();
        VcudaDeviceSynchronize();
    }

    // sam:
    if (UsingLightSource && TurnOnLightAfterThisManySeconds < currentTime) {
        LightAPos = VX3_Vec3D<>(LightAPosX*voxSize, LightAPosY*voxSize, LightAPosZ*voxSize);
        LightBPos = VX3_Vec3D<>(LightBPosX*voxSize, LightBPosY*voxSize, LightBPosZ*voxSize);
        int lightOn = 0;
        if (VX3_MathTree::eval(0, 0, 0, 0, currentTime, 0, 0, 0, 0, lightA_function) > 0 )
            lightOn += 1;
        if (VX3_MathTree::eval(0, 0, 0, 0, currentTime, 0, 0, 0, 0, lightB_function) > 0 )
            lightOn += 2;
        updateOcclusion(lightOn);
    }

    //sam:
    if (EnableDisintegration) {
        updateVoxelDetachment();  // find voxels to break off
        updateDetach(); // cut links
    }

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_voxels, 0,
                                       num_d_voxels); // Dynamically calculate blockSize
    int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize;
    int blockSize_voxels = num_d_voxels < blockSize ? num_d_voxels : blockSize;
    gpu_update_voxels<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, num_d_voxels, dt, currentTime, this);
    CUDA_CHECK_AFTER_CALL();
    VcudaDeviceSynchronize();

    int CycleStep =
        int(TempPeriod / dt); // Sample at the same time point in the cycle, to avoid the impact of actuation as much as possible.
    if (CurStepCount % CycleStep == 0) {
        angleSampleTimes++;

        currentCenterOfMass_history[0] = currentCenterOfMass_history[1];
        currentCenterOfMass_history[1] = currentCenterOfMass;
        updateCurrentCenterOfMass();
        auto A = currentCenterOfMass_history[0];
        auto B = currentCenterOfMass_history[1];
        auto C = currentCenterOfMass;
        if (B == C || A == B || angleSampleTimes < 3) {
            recentAngle = 0; // avoid divide by zero, and don't include first two steps where A and B are still 0.
        } else {
            recentAngle = acos((B - A).Dot(C - B) / (B.Dist(A) * C.Dist(B)));
        }
        // printf("(%d) recentAngle = %f\n", angleSampleTimes, recentAngle);

        // Also calculate targetCloseness here.
        computeTargetCloseness();
    }

    if (SecondaryExperiment) {
        // SecondaryExperiment handle tags:
        // RemoveFromSimulationAfterThisManySeconds
        // ReinitializeInitialPositionAfterThisManySeconds
        // TurnOnThermalExpansionAfterThisManySeconds
        // TurnOnCiliaAfterThisManySeconds

        removeVoxels();
        if (InitialPositionReinitialized == false && ReinitializeInitialPositionAfterThisManySeconds < currentTime) {
            InitialPositionReinitialized = true;
            InitializeCenterOfMass();
            saveInitialPosition();
        }

    }

    currentTime += dt;
    // time_measures[1] = clock();
    // printf("running time for each step: \n");
    // for (int i=0;i<1;i++)
    //     printf("\t%d) %ld clock cycles.\n", i,
    //     time_measures[i+1]-time_measures[i]);
    return true;
}

__device__ void VX3_VoxelyzeKernel::InitializeCenterOfMass() {
    initialCenterOfMass = currentCenterOfMass;
}

__device__ void VX3_VoxelyzeKernel::removeVoxels() {
    for (int i=0;i<num_d_voxelMats;i++) {
        if (d_voxelMats[i].removed == false &&
        d_voxelMats[i].RemoveFromSimulationAfterThisManySeconds > 0 &&
        d_voxelMats[i].RemoveFromSimulationAfterThisManySeconds < currentTime ) {
            VX3_Voxel* neighbor_voxel;

            for (int j=0;j<num_d_voxels;j++) {
                if (d_voxels[j].mat == &d_voxelMats[i] && d_voxels[j].removed == false) {
                    d_voxels[j].removed = true; // mark this voxel as removed
                    for (int k=0;k<6;k++) { // check links in all direction
                        if (d_voxels[j].links[k]) {
                            d_voxels[j].links[k]->removed = true; // mark the link as removed
                            if (d_voxels[j].links[k]->pVNeg == &d_voxels[j]) { // this voxel is pVNeg
                                neighbor_voxel = d_voxels[j].links[k]->pVPos;
                            } else {
                                neighbor_voxel = d_voxels[j].links[k]->pVNeg;
                            }
                            for (int m=0;m<6;m++) {
                                if (neighbor_voxel->links[m] == d_voxels[j].links[k]) {
                                    neighbor_voxel->links[m] = NULL; // delete the neighbor's link
                                    break;
                                }
                            }
                            d_voxels[j].links[k] = NULL; // delete this voxel's link
                        }
                    }
                }
            }
            d_voxelMats[i].removed = true;
            isSurfaceChanged = true;
        }
    }

}

__device__ void VX3_VoxelyzeKernel::updateAttach() {
    // for each surface voxel pair, check distance < watchDistance, make a new
    // link between these two voxels, updateSurface().
    int blockSize;
    int minGridSize;
    if (false) {
        // the parameters of grid are set in gpu_update_voxels, so detection only useful after initialization
        if (gridLowerBound != gridUpperBound) {
            gridDelta = (gridUpperBound - gridLowerBound) / lookupGrid_n;
            if (gridDelta.x < voxSize * 2) {
                gridDelta.x = voxSize * 2;
            }
            if (gridDelta.y < voxSize * 2) {
                gridDelta.y = voxSize * 2;
            }
            if (gridDelta.z < voxSize * 2) {
                gridDelta.z = voxSize * 2;
            }
            // printf("gridLowerBound (%f,%f,%f), gridDelta (%f,%f,%f), gridUpperBound (%f,%f,%f).\n\n", gridLowerBound.x, gridLowerBound.y,
            //        gridLowerBound.z, gridDelta.x, gridDelta.y, gridDelta.z, gridUpperBound.x, gridUpperBound.y, gridUpperBound.z);
            // clear all lookupGrids
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_clear_lookupgrid, 0,
                                               num_lookupGrids); // Dynamically calculate blockSize
            int gridSize_voxels = (num_lookupGrids + blockSize - 1) / blockSize;
            int blockSize_voxels = num_lookupGrids < blockSize ? num_lookupGrids : blockSize;
            gpu_clear_lookupgrid<<<gridSize_voxels, blockSize_voxels>>>(d_collisionLookupGrid, num_lookupGrids);
            CUDA_CHECK_AFTER_CALL();
            VcudaDeviceSynchronize();
            // build lookupGrids: put surface voxels into grids
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_insert_lookupgrid, 0,
                                               num_d_surface_voxels); // Dynamically calculate blockSize
            gridSize_voxels = (num_d_surface_voxels + blockSize - 1) / blockSize;
            blockSize_voxels = num_d_surface_voxels < blockSize ? num_d_surface_voxels : blockSize;
            gpu_insert_lookupgrid<<<gridSize_voxels, blockSize_voxels>>>(d_surface_voxels, num_d_surface_voxels, d_collisionLookupGrid,
                                                                         &gridLowerBound, &gridDelta, lookupGrid_n);
            CUDA_CHECK_AFTER_CALL();
            VcudaDeviceSynchronize();
            // detect collision: voxels in each grid with voxels within this grid and its neighbors
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_collision_attachment_lookupgrid, 0,
                                               num_lookupGrids); // Dynamically calculate blockSize
            gridSize_voxels = (num_lookupGrids + blockSize - 1) / blockSize;
            blockSize_voxels = num_lookupGrids < blockSize ? num_lookupGrids : blockSize;
            gpu_collision_attachment_lookupgrid<<<gridSize_voxels, blockSize_voxels>>>(d_collisionLookupGrid, num_lookupGrids,
                                                                                       watchDistance, this);
            CUDA_CHECK_AFTER_CALL();
            VcudaDeviceSynchronize();
        }
    } else {
        // Pairwise detection O(n ^ 2)
        blockSize = 16;
        dim3 dimBlock(blockSize, blockSize);
        dim3 dimGrid((num_d_surface_voxels + dimBlock.x - 1) / dimBlock.x, (num_d_surface_voxels + dimBlock.y - 1) / dimBlock.y);
        // printf("num_d_surface_voxels %d\n", num_d_surface_voxels);
        gpu_update_attach<<<dimGrid, dimBlock>>>(d_surface_voxels, num_d_surface_voxels, watchDistance,
                                                 this); // invoke two dimensional gpu threads 'CUDA C++ Programming
                                                        // Guide', Nov 2019, P52.
        CUDA_CHECK_AFTER_CALL();
    }
}

__device__ void VX3_VoxelyzeKernel::updateDetach() {
    if (d_v_links.size()) {
        int minGridSize, blockSize;
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_detach, 0,
                                           d_v_links.size()); // Dynamically calculate blockSize
        int gridSize_links = (d_v_links.size() + blockSize - 1) / blockSize;
        int blockSize_links = d_v_links.size() < blockSize ? d_v_links.size() : blockSize;
        // if (CurStepCount % 1000 == 0 || currentTime>1.0) {
        //     printf("&d_v_links[0] %p; d_v_links.size() %d. \n", &d_v_links[0], d_v_links.size());
        // }
        gpu_update_detach<<<gridSize_links, blockSize_links>>>(&d_v_links[0], d_v_links.size(), this);
        CUDA_CHECK_AFTER_CALL();
        VcudaDeviceSynchronize();
    }
}

// sam:
__device__ void VX3_VoxelyzeKernel::updateOcclusion(int lightOn) {
    int minGridSize, blockSize;
    if (OnlySurfVoxOcclude) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_occlusion, 0, num_d_surface_voxels);
        int gridSize_voxels = (num_d_surface_voxels + blockSize - 1) / blockSize;
        int blockSize_voxels = num_d_surface_voxels < blockSize ? num_d_surface_voxels : blockSize;
        gpu_update_occlusion<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, d_surface_voxels, num_d_surface_voxels, this, true, lightOn);
    }
    else {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_occlusion, 0, num_d_voxels);
        int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize;
        int blockSize_voxels = num_d_voxels < blockSize ? num_d_voxels : blockSize;
        gpu_update_occlusion<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, d_surface_voxels, num_d_voxels, this, false, lightOn);
    }
    CUDA_CHECK_AFTER_CALL();
    VcudaDeviceSynchronize();
}

// sam:
__device__ void VX3_VoxelyzeKernel::updateVoxelDetachment() {
    int minGridSize, blockSize;
    if (UsingLightSource) {  // make a tag for this? or just assume lightsource + disintegrate = laser
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_voxel_detachment, 0, num_d_surface_voxels);
        int gridSize_voxels = (num_d_surface_voxels + blockSize - 1) / blockSize;
        int blockSize_voxels = num_d_surface_voxels < blockSize ? num_d_surface_voxels : blockSize;
        gpu_update_voxel_detachment<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, d_surface_voxels, num_d_surface_voxels, this, true);
    }
    else {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_occlusion, 0, num_d_voxels);
        int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize;
        int blockSize_voxels = num_d_voxels < blockSize ? num_d_voxels : blockSize;
        gpu_update_voxel_detachment<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, d_surface_voxels, num_d_voxels, this, false);
    }
    CUDA_CHECK_AFTER_CALL();
    VcudaDeviceSynchronize();
}

__device__ void VX3_VoxelyzeKernel::updateCurrentCenterOfMass() {
    double TotalMass = 0;
    VX3_Vec3D<> Sum(0, 0, 0);
    for (int i = 0; i < num_d_voxels; i++) {
        if (!d_voxels[i].mat->isMeasured || d_voxels[i].removed) {  // sam: || d_voxels[i].removed
            continue;
        }
        double ThisMass = d_voxels[i].material()->mass();
        Sum += d_voxels[i].position() * ThisMass;
        TotalMass += ThisMass;
    }
    if (TotalMass==0) {
        currentCenterOfMass = VX3_Vec3D<>();
        return;
    }
    currentCenterOfMass = Sum / TotalMass;
}

__device__ void VX3_VoxelyzeKernel::regenerateSurfaceVoxels() {
    // regenerate d_surface_voxels
    if (d_surface_voxels) {
        delete d_surface_voxels;
        d_surface_voxels = NULL;
    }
    VX3_dVector<VX3_Voxel *> tmp;
    for (int i = 0; i < num_d_voxels; i++) {
        d_voxels[i].updateSurface();
        if (d_voxels[i].isSurface() && !d_voxels[i].removed) {
            tmp.push_back(&d_voxels[i]);
        }
    }
    num_d_surface_voxels = tmp.size();
    d_surface_voxels = (VX3_Voxel **)malloc(num_d_surface_voxels * sizeof(VX3_Voxel));
    for (int i = 0; i < num_d_surface_voxels; i++) {
        d_surface_voxels[i] = tmp[i];
    }
}

__device__ VX3_MaterialLink *VX3_VoxelyzeKernel::combinedMaterial(VX3_MaterialVoxel *mat1, VX3_MaterialVoxel *mat2) {
    for (int i = 0; i < d_v_linkMats.size(); i++) {
        VX3_MaterialLink *thisMat = d_v_linkMats[i];
        if ((thisMat->vox1Mat == mat1 && thisMat->vox2Mat == mat2) || (thisMat->vox1Mat == mat2 && thisMat->vox2Mat == mat1))
            return thisMat; // already exist
    }

    VX3_MaterialLink *newMat = new VX3_MaterialLink(mat1, mat2); // where to free this?
    d_v_linkMats.push_back(newMat);
    mat1->d_dependentMaterials.push_back(newMat);
    mat2->d_dependentMaterials.push_back(newMat);

    return newMat;
}

__device__ void VX3_VoxelyzeKernel::computeFitness() {
    VX3_Vec3D<> offset = currentCenterOfMass - initialCenterOfMass;
    fitness_score = VX3_MathTree::eval(offset.x, offset.y, offset.z, collisionCount, currentTime, recentAngle, targetCloseness,
                                       numClosePairs, num_d_voxels, fitness_function);
}

__device__ void VX3_VoxelyzeKernel::registerTargets() {
    for (int i = 0; i < num_d_voxels; i++) {
        auto v = &d_voxels[i];
        if (v->mat->isTarget) {
            d_targets.push_back(v);
        }
    }
}

__device__ void VX3_VoxelyzeKernel::computeTargetCloseness() {
    // this function is called periodically. not very often. once every thousands of steps.
    if (MaxDistInVoxelLengthsToCountAsPair==0)
        return;
    double R = MaxDistInVoxelLengthsToCountAsPair * voxSize;
    double ret = 0;
    numClosePairs = 0;
    for (int i = 0; i < d_targets.size(); i++) {
        for (int j = i + 1; j < d_targets.size(); j++) {
            double distance = d_targets[i]->pos.Dist(d_targets[j]->pos);
            if (distance < R) {
                numClosePairs++;
            }
            ret += 1 / distance;
        }
    }
    targetCloseness = ret;
    // printf("targetCloseness: %f\n", targetCloseness);
}

/* Sub GPU Threads */
__global__ void gpu_update_links(VX3_Link **links, int num) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        VX3_Link *t = links[gindex];
        if (t->removed)
            return;
        if (t->pVPos->mat->fixed && t->pVNeg->mat->fixed)
            return;
        if (t->isDetached)
            return;
        t->updateForces();
        if (t->axialStrain() > 100) {
            printf(COLORCODE_BOLD_RED "ERROR: Diverged.");
        }
    }
}
__global__ void gpu_update_voxels(VX3_Voxel *voxels, int num, double dt, double currentTime, VX3_VoxelyzeKernel *k) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        VX3_Voxel *t = &voxels[gindex];
        if (t->removed)
            return;
        if (t->mat->fixed)
            return; // fixed voxels, no need to update position
        t->timeStep(dt, currentTime, k);

        // update lower bound and upper bound
        if (t->pos.x < k->gridLowerBound.x) {
            k->gridLowerBound.x = t->pos.x;
        } else if (t->pos.x > k->gridUpperBound.x) {
            k->gridUpperBound.x = t->pos.x;
        }
        if (t->pos.y < k->gridLowerBound.y) {
            k->gridLowerBound.y = t->pos.y;
        } else if (t->pos.y > k->gridUpperBound.y) {
            k->gridUpperBound.y = t->pos.y;
        }
        if (t->pos.z < k->gridLowerBound.z) {
            k->gridLowerBound.z = t->pos.z;
        } else if (t->pos.z > k->gridUpperBound.z) {
            k->gridUpperBound.z = t->pos.z;
        }
        // update sticky status
        t->enableAttach = false;
        if (VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[0]) > 0 &&
            VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[1]) > 0 &&
            VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[2]) > 0 &&
            VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[3]) > 0 &&
            VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[4]) > 0) {
            t->enableAttach = true;
        };
    }
}

__global__ void gpu_update_temperature(VX3_Voxel *voxels, int num, double TempAmplitude, double TempPeriod, double currentTime, VX3_VoxelyzeKernel* k) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        // vfloat tmp = pEnv->GetTempAmplitude() *
        // sin(2*3.1415926f*(CurTime/pEnv->GetTempPeriod() + pV->phaseOffset)) -
        // pEnv->GetTempBase();
        VX3_Voxel *t = &voxels[gindex];
        if (t->removed)
            return;
        if (t->mat->TurnOnThermalExpansionAfterThisManySeconds > currentTime)
            return;
        if (t->mat->fixed)
            return; // fixed voxels, no need to update temperature
        // // sam:
        // if (t->isDetached)
        //     return; 
        double currentTemperature =
            TempAmplitude * sin(2 * 3.1415926f * (currentTime / TempPeriod + t->phaseOffset)); // update the global temperature
        // TODO: if we decide not to use PhaseOffset any more, we can move this calculation outside.
        // By default we don't enable expansion. But we can enable that in VXA.
        if (!k->EnableExpansion) {
            if (currentTemperature > 0) {
                currentTemperature = 0;
            }
        }
        t->setTemperature(currentTemperature);
        // t->setTemperature(0.0f);
    }
}
__device__ bool is_neighbor(VX3_Voxel *voxel1, VX3_Voxel *voxel2, VX3_Link *incoming_link, int depth) {
    // printf("Checking (%d,%d,%d) and (%d,%d,%d) in depth %d.\n",
    //             voxel1->ix, voxel1->iy, voxel1->iz,
    //             voxel2->ix, voxel2->iy, voxel2->iz, depth);
    if (voxel1 == voxel2) {
        // printf("found.\n");
        return true;
    }
    if (depth <= 0) { // cannot find in depth
        // printf("not found.\n");
        return false;
    }
    for (int i = 0; i < 6; i++) {
        if (voxel1->links[i]) {
            if (voxel1->links[i] != incoming_link) {
                if (voxel1->links[i]->pVNeg == voxel1) {
                    if (is_neighbor(voxel1->links[i]->pVPos, voxel2, voxel1->links[i], depth - 1)) {
                        return true;
                    }
                } else {
                    if (is_neighbor(voxel1->links[i]->pVNeg, voxel2, voxel1->links[i], depth - 1)) {
                        return true;
                    }
                }
            }
        }
    }
    // printf("not found.\n");
    return false;
}

__device__ void handle_collision_attachment(VX3_Voxel *voxel1, VX3_Voxel *voxel2, double watchDistance, VX3_VoxelyzeKernel *k) {
    // if both of the voxels are fixed, no need to compute.
    if (voxel1->mat->fixed && voxel2->mat->fixed)
        return;

    VX3_Vec3D<double> diff = voxel1->pos - voxel2->pos;
    watchDistance = (voxel1->baseSizeAverage() + voxel2->baseSizeAverage()) * COLLISION_ENVELOPE_RADIUS;

    if (diff.x > watchDistance || diff.x < -watchDistance)
        return;
    if (diff.y > watchDistance || diff.y < -watchDistance)
        return;
    if (diff.z > watchDistance || diff.z < -watchDistance)
        return;

    if (diff.Length() > watchDistance)
        return;

    // to exclude voxels already have link between them. check in depth of
    // 1, direct neighbor ignore the collision
    if (is_neighbor(voxel1, voxel2, NULL, 1)) {
        return;
    }
    // calculate and store contact force, apply and clean in
    // VX3_Voxel::force()
    // if (voxel1->mat !=
    //     voxel2->mat) { // disable same material collision for now
    VX3_Vec3D<> cache_contactForce1, cache_contactForce2;
    if (k->EnableCollision) {
        VX3_Collision collision(voxel1, voxel2);
        collision.updateContactForce();
        cache_contactForce1 = collision.contactForce(voxel1);
        cache_contactForce2 = collision.contactForce(voxel2);
        voxel1->contactForce += cache_contactForce1;
        voxel2->contactForce += cache_contactForce2;
        if ((voxel1->mat->isTarget && !voxel2->mat->isTarget) || (voxel2->mat->isTarget && !voxel1->mat->isTarget)) {
            atomicAdd(&k->collisionCount, 1);
            if (k->EnableSignals) {
                if (voxel1->mat->isTarget) {
                    voxel2->receiveSignal(100, k->currentTime, true);
                } else {
                    voxel1->receiveSignal(100, k->currentTime, true);
                }
            }
        }
    }

    // determined by formula
    if (!voxel1->enableAttach || !voxel2->enableAttach)
        return;

    // fixed voxels, no need to look further for attachment
    if (voxel1->mat->fixed || voxel2->mat->fixed)
        return;
    // different material, no need to attach
    if (voxel1->mat != voxel2->mat)
        return;
    if (!voxel1->mat->sticky)
        return;

    // to exclude voxels already have link between them. check in depth 5.
    // closely connected part ignore the link creation.
    if (is_neighbor(voxel1, voxel2, NULL, 5)) {
        return;
    }

    // determine relative position
    linkDirection link_dir_1, link_dir_2;
    linkAxis link_axis;
    auto a = voxel1->orientation();
    auto b = voxel2->orientation();
    auto c = voxel1->position();
    auto d = voxel2->position();
    auto e = c - d;
    auto ea = a.RotateVec3DInv(-e);
    auto eb = b.RotateVec3DInv(e);

    // first find which is the dominant axis, then determine which one is
    // neg which one is pos.
    VX3_Vec3D<double> f;
    bool reverseOrder = false;
    f = ea.Abs();
    if (f.x >= f.y && f.x >= f.z) { // X_AXIS
        link_axis = X_AXIS;
        if (ea.x < 0) {
            link_dir_1 = X_NEG;
            link_dir_2 = X_POS;
            reverseOrder = true;
        } else {
            link_dir_1 = X_POS;
            link_dir_2 = X_NEG;
        }
    } else if (f.y >= f.x && f.y >= f.z) { // Y_AXIS
        link_axis = Y_AXIS;
        if (ea.y < 0) {
            link_dir_1 = Y_NEG;
            link_dir_2 = Y_POS;
            reverseOrder = true;
        } else {
            link_dir_1 = Y_POS;
            link_dir_2 = Y_NEG;
        }
    } else { // Z_AXIS
        link_axis = Z_AXIS;
        if (ea.z < 0) { // voxel1 is on top
            link_dir_1 = Z_NEG;
            link_dir_2 = Z_POS;
            reverseOrder = true;
        } else {
            link_dir_1 = Z_POS;
            link_dir_2 = Z_NEG;
        }
    }

    // TODO: need to solve this. Create only when there's a right place to
    // attach
    if (voxel1->links[link_dir_1] == NULL && voxel2->links[link_dir_2] == NULL) {
        VX3_Link *pL;
        if (reverseOrder) {
            pL = new VX3_Link(voxel1, link_dir_1, voxel2, link_dir_2, link_axis,
                              k); // make the new link (change to both materials, etc.
        } else {
            pL = new VX3_Link(voxel2, link_dir_2, voxel1, link_dir_1, link_axis,
                              k); // make the new link (change to both materials, etc.
        }
        if (!pL) {
            printf(COLORCODE_BOLD_RED "ERROR: Out of memory. Link not created.\n");
            return;
        }
        pL->isNewLink = k->SafetyGuard;
        k->d_v_links.push_back(pL); // add to the list

        k->isSurfaceChanged = true;

        // printf("createLink.... %p %p distance=> %f %f %f (%f), dir (%d and "
        //        "%d), watchDistance %f.\n",
        //        voxel1, voxel2, diff.x, diff.y, diff.z, diff.Length(),
        //        link_dir_1, link_dir_2, watchDistance);
        // printf("orientation (%f; %f, %f, %f) and (%f; %f, %f, %f).\n", a.w,
        //        a.x, a.y, a.z, b.w, b.x, b.y, b.z);
        // printf("ea, after inv rotate (%f, %f, %f)", ea.x, ea.y, ea.z);
        // printf("newLink: rest %f.\n", pL->currentRestLength);
        // printf("between (%d,%d,%d) and (%d,%d,%d).\n", voxel1->ix,
        //        voxel1->iy, voxel1->iz, voxel2->ix, voxel2->iy, voxel2->iz);

        // if a link is created, set contact force = 0 , for stable reason. (if they are connected, they should not collide.)
        voxel1->contactForce -= cache_contactForce1;
        voxel2->contactForce -= cache_contactForce2;
    }
}

__global__ void gpu_update_attach(VX3_Voxel **surface_voxels, int num, double watchDistance, VX3_VoxelyzeKernel *k) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int second = threadIdx.y + blockIdx.y * blockDim.y;
    if (first < num && second < first) {
        VX3_Voxel *voxel1 = surface_voxels[first];
        VX3_Voxel *voxel2 = surface_voxels[second];
        if (voxel1->removed || voxel2->removed)
            return;
        handle_collision_attachment(voxel1, voxel2, watchDistance, k);
    }
}

__global__ void gpu_update_cilia_force(VX3_Voxel **surface_voxels, int num, VX3_VoxelyzeKernel *k) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < num) {
        if (surface_voxels[index]->removed)
            return;
        if (surface_voxels[index]->mat->Cilia == 0)
            return;
        if (surface_voxels[index]->mat->TurnOnCiliaAfterThisManySeconds > k->currentTime)
            return;
        // rotate base cilia force and update it into voxel.

        // sam:
        if (k->UsingLightSource) {
            surface_voxels[index]->CiliaForce = surface_voxels[index]->orient.RotateVec3D(surface_voxels[index]->baseCiliaForce);
            VX3_Vec3D<double> force = surface_voxels[index]->CiliaForce;
            double light = surface_voxels[index]->lightStored / k->LightSensitiveTime;  // in [0,1]
            double effect = k->CiliaFactorInLight;
            // note: we can now use per vox sensitivity: surface_voxels[index]->photosensitivity
            if (k->UsingVolvox && light > 0)
                surface_voxels[index]->CiliaForce += (1 - light) * (force*effect - force); // volvox get full effect then decay
            else
                surface_voxels[index]->CiliaForce += light * (force*effect - force);  // add accumulated light effect to cilia force
        }

        else {
            surface_voxels[index]->CiliaForce = surface_voxels[index]->orient.RotateVec3D(
                surface_voxels[index]->baseCiliaForce + surface_voxels[index]->localSignal * surface_voxels[index]->shiftCiliaForce);
        }
    }
}

// sam:
__global__ void gpu_update_occlusion(VX3_Voxel *voxels, VX3_Voxel **surface_voxels, int num, VX3_VoxelyzeKernel *k, bool surfVoxOnly, int lightOn) {
    // https://gamedev.stackexchange.com/questions/18436/most-efficient-aabb-vs-ray-collision-algorithms

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (index < num) {

        VX3_Voxel *thisVox = &voxels[index];
        if (surfVoxOnly)
            thisVox = surface_voxels[index];

        if (thisVox->removed) {
            return;
        }

        if (lightOn == 0) { // then everything is in the dark
            thisVox->inShade = true;

            if (thisVox->mat->isLightSourceA || thisVox->mat->isLightSourceB){
                thisVox->localSignal = 0;
                return;
            }

            if (thisVox->lightStored > 0)
                thisVox->lightStored -= 1;

            if (k->UsingVolvox)
                thisVox->localSignal = 1 - thisVox->lightStored / k->LightSensitiveTime;
            else
                thisVox->localSignal = thisVox->lightStored / k->LightSensitiveTime;

            if (k->UsingVolvox && thisVox->lightStored == 0)
                thisVox->localSignal = 0; // just for drawing

            return;
        }

        if (lightOn == 3) { // ASSUMPTION: everything is in the light
            thisVox->inShade = true;

            if (thisVox->mat->isLightSourceA || thisVox->mat->isLightSourceB){
                thisVox->localSignal = 1;
                return;
            }

            if (thisVox->lightStored > 0)
                thisVox->lightStored += 1;

            if (k->UsingVolvox)
                thisVox->localSignal = 1 - thisVox->lightStored / k->LightSensitiveTime;
            else
                thisVox->localSignal = thisVox->lightStored / k->LightSensitiveTime;

            if (k->UsingVolvox && thisVox->lightStored == 0)
                thisVox->localSignal = 0; // just for drawing

            return;
        }

        if (lightOn == 1 && thisVox->mat->isLightSourceA) {
            thisVox->localSignal = 1;
            return;
        }

        if (lightOn == 1 && thisVox->mat->isLightSourceB) {
            thisVox->localSignal = 0;
            return;
        }

        if (lightOn == 2 && thisVox->mat->isLightSourceA) {
            thisVox->localSignal = 0;
            return;
        }

        if (lightOn == 2 && thisVox->mat->isLightSourceB) {
            thisVox->localSignal = 1;
            return;
        }

        if (!thisVox->mat->lightSensitive) {
            return;
        }
            
        // double prevTimeInDark = thisVox->timeInDark;
        // double prevTimeInLight = thisVox->timeInLight;

        thisVox->inShade = false;

        VX3_Vec3D<double> ray_origin = thisVox->position();

        // TODO: only one can be on at a time or else light B overrides
        VX3_Vec3D<> LightPos;
        if (lightOn == 1)
            LightPos = k->LightAPos;
        if (lightOn == 2)
            LightPos = k->LightBPos; 

        for (int j = 0; j < num; j++) {

            if (j == index)
                continue;
            
            // does the ray from thisVox to k->LightPos intersect with otherVox's bounding box?
            VX3_Voxel *otherVox = &voxels[j];
            if (surfVoxOnly)
                otherVox = surface_voxels[j];

            if (otherVox->mat->transparent || otherVox->isDetached || otherVox->removed)  // todo: detached don't occlude tag
                continue;

            // lb is the corner of AABB with minimal coordinates - left bottom, rt is maximal corner
            VX3_Vec3D<double> lb = otherVox->position() + otherVox->cornerOffset(NNN);
            VX3_Vec3D<double> rt = otherVox->position() + otherVox->cornerOffset(PPP);

            // vector from this voxel to other voxel 
            VX3_Vec3D<double> thisVoxToOtherVox = otherVox->position() - ray_origin; // ray_origin ---> otherVox origin
            VX3_Vec3D<double> thisVoxToLight = LightPos - ray_origin ;  // ray_origin ---> k->LightPos  // apply inverse square law?

            // can't occlude on far side of the light source
            if (thisVoxToOtherVox.Length2() > thisVoxToLight.Length2())
                continue;

            // unit direction vector of ray
            VX3_Vec3D<double> unitdir = thisVoxToLight.Normalized();
            
            // // add a tiny bit so we don't divide by zero in the next step? does this ever happen?
            // unitdir.x = unitdir.x == 0 ? 1e-10 : unitdir.x;
            // unitdir.y = unitdir.y == 0 ? 1e-10 : unitdir.y;
            // unitdir.z = unitdir.z == 0 ? 1e-10 : unitdir.z;

            VX3_Vec3D<float> dirfrac;
            dirfrac.x = 1.0f / unitdir.x;
            dirfrac.y = 1.0f / unitdir.y;
            dirfrac.z = 1.0f / unitdir.z;

            float t1 = (lb.x - ray_origin.x)*dirfrac.x;
            float t2 = (rt.x - ray_origin.x)*dirfrac.x;
            float t3 = (lb.y - ray_origin.y)*dirfrac.y;
            float t4 = (rt.y - ray_origin.y)*dirfrac.y;
            float t5 = (lb.z - ray_origin.z)*dirfrac.z;
            float t6 = (rt.z - ray_origin.z)*dirfrac.z;

            float tmin = max(max(min(t1, t2), min(t3, t4)), min(t5, t6));
            float tmax = min(min(max(t1, t2), max(t3, t4)), max(t5, t6));
            
            // float t;

            // if tmax < 0, ray (line) is intersecting AABB, but the whole AABB is behind us
            if (tmax < 0)
            {
                // t = tmax;
                continue;
            }

            // if tmin > tmax, ray doesn't intersect AABB
            if (tmin > tmax)
            {
                // t = tmax;
                continue;
            }

            // t = tmin;
            thisVox->inShade = true;
            if (thisVox->lightStored > 0)
                thisVox->lightStored -= 1;
            break;
        }
        // done checking for occlusion here
        if (!thisVox->inShade) {
            if (thisVox->lightStored < k->LightSensitiveTime)
                thisVox->lightStored += 1;
        }
        // for drawing
        if (k->UsingVolvox && thisVox->lightStored>0)
            thisVox->localSignal = 1 - thisVox->lightStored / k->LightSensitiveTime;
        else
            thisVox->localSignal = thisVox->lightStored / k->LightSensitiveTime;
    }
}

__global__ void gpu_clear_lookupgrid(VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid, int num) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < num) {
        d_collisionLookupGrid[index].clear();
    }
}

__global__ void gpu_insert_lookupgrid(VX3_Voxel **d_surface_voxels, int num, VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid,
                                      VX3_Vec3D<> *gridLowerBound, VX3_Vec3D<> *gridDelta, int lookupGrid_n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < num) {
        VX3_Voxel *v = d_surface_voxels[index];
        int ix = int((v->pos.x - gridLowerBound->x) / gridDelta->x);
        int iy = int((v->pos.y - gridLowerBound->y) / gridDelta->y);
        int iz = int((v->pos.z - gridLowerBound->z) / gridDelta->z);
        bound(ix, 0, lookupGrid_n);
        bound(iy, 0, lookupGrid_n);
        bound(iz, 0, lookupGrid_n);
        d_collisionLookupGrid[ix * lookupGrid_n * lookupGrid_n + iy * lookupGrid_n + iz].push_back(v);
    }
}

__global__ void gpu_pairwise_detection(VX3_Voxel **voxel1, VX3_Voxel **voxel2, int num_v1, int num_v2, double watchDistance,
                                       VX3_VoxelyzeKernel *k) {
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    if (index_x < num_v1 && index_y < num_v2) {
        if (voxel1[index_x]->removed || voxel2[index_y]->removed)
            return;
        handle_collision_attachment(voxel1[index_x], voxel2[index_y], watchDistance, k);
    }
}

__device__ int index_3d_to_1d(int x, int y, int z, int dim_len) { return x * dim_len * dim_len + y * dim_len + z; }
__device__ VX3_Vec3D<int> index_1d_to_3d(int n, int dim_len) {
    VX3_Vec3D<int> v;
    v.x = int(floor(double(n / (dim_len * dim_len)))) % dim_len;
    v.y = int(floor(double(n / dim_len))) % dim_len;
    v.z = n % dim_len;
    return v;
}

__global__ void gpu_collision_attachment_lookupgrid(VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid, int num, double watchDistance,
                                                    VX3_VoxelyzeKernel *k) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < num) {
        int num_voxel_in_grid = d_collisionLookupGrid[index].size();
        if (num_voxel_in_grid == 0)
            return;
        // within the grid
        int dim_len = k->lookupGrid_n;
        auto index_3d = index_1d_to_3d(index, dim_len);
        int ix = index_3d.x;
        int iy = index_3d.y;
        int iz = index_3d.z;
        // printf("num_voxel_in_grid %d[%d][%d][%d]: %d\n", index, ix, iy, iz, num_voxel_in_grid);
        int blockSize = 16;
        dim3 dimBlock(blockSize, blockSize);
        dim3 dimGrid((num_voxel_in_grid + dimBlock.x - 1) / dimBlock.x, (num_voxel_in_grid + dimBlock.y - 1) / dimBlock.y);
        gpu_pairwise_detection<<<dimGrid, dimBlock>>>(&d_collisionLookupGrid[index][0], &d_collisionLookupGrid[index][0], num_voxel_in_grid,
                                                      num_voxel_in_grid, watchDistance, k);
        // invoke two dimensional gpu threads 'CUDA C++ Programming
        // Guide', Nov 2019, P52.
        CUDA_CHECK_AFTER_CALL();
        // with neighbors
        for (int dix = -1; dix <= 1; dix++) {
            for (int diy = -1; diy <= 1; diy++) {
                for (int diz = -1; diz <= 1; diz++) {
                    int index_2 = index_3d_to_1d(ix + dix, iy + diy, iz + diz, dim_len);
                    if (index_2 > index && index_2 < num) {
                        int num_voxel_in_grid_2 = d_collisionLookupGrid[index_2].size();
                        if (num_voxel_in_grid_2 > 0) {
                            gpu_pairwise_detection<<<dimGrid, dimBlock>>>(
                                &d_collisionLookupGrid[index][0],
                                &d_collisionLookupGrid[index_3d_to_1d(ix + dix, iy + diy, iz + diz, dim_len)][0], num_voxel_in_grid,
                                num_voxel_in_grid_2, watchDistance, k);
                        }
                    }
                }
            }
        }
        CUDA_CHECK_AFTER_CALL();
    }
}

__global__ void gpu_update_detach(VX3_Link **links, int num, VX3_VoxelyzeKernel* k) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        VX3_Link *t = links[gindex];
        if (t->removed)
            return;
        if (t->isDetached)
            return;
        // clu: vxa: MatModel=1, Fail_Stress=1e+6 => Fail_Stress => failureStress => isFailed.
        if (t->isFailed() || t->detachMe) {
            t->isDetached = true;
            t->removed = true;
            for (int i = 0; i < 6; i++) {
                if (t->pVNeg->links[i] == t) {
                    t->pVNeg->links[i] = NULL;
                }
                if (t->pVPos->links[i] == t) {
                    t->pVPos->links[i] = NULL;
                }
            }
            k->isSurfaceChanged = true;
        }
    }
}

// sam:
__global__ void gpu_update_voxel_detachment(VX3_Voxel *voxels, VX3_Voxel **surface_voxels, int num, VX3_VoxelyzeKernel* k, bool surfVoxOnly) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < num) {
        
        VX3_Voxel *thisVox = &voxels[index];
        if (surfVoxOnly)
            thisVox = surface_voxels[index];

        if (thisVox->removed)
            return;
        // if (thisVox->isDetached)
        //     return;
        if (thisVox->mat->fixed)
            return;

        if (!thisVox->mat->detachable)
            return;
        
        if (!k->UsingLightSource && thisVox->detachTime > 0 && k->currentTime <= thisVox->detachTime) { 
            thisVox->localSignal = k->currentTime / thisVox->detachTime;
        }
        
        else if (thisVox->localSignal >= 1 || k->currentTime >= thisVox->detachTime) {
            thisVox->isDetached = true;
            thisVox->removed = true;
            for (int k=0;k<6;k++) { // check links in all direction
                if (thisVox->links[k]) {
                    thisVox->links[k]->detachMe = true;
                }
            }
        }
    }
}